#include "hip/hip_runtime.h"
#include <gputk.h>
#include <algorithm>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#define BLOCK_SIZE 1024
#define THRESHOLD (BLOCK_SIZE * 32)

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

struct pt_idx {
  unsigned idx;
  float x, y;

  bool operator==(const pt_idx& rhs) const
  {
    return x == rhs.x && y == rhs.y && idx == rhs.idx;
  }
};

int ccw(pt_idx p1, pt_idx p2, pt_idx p)
{
  float prod = (p2.x - p1.x) * (p.y - p1.y) - (p2.y - p1.y) * (p.x - p1.x);
  if (prod > 0)
  {
    return 1;
  }
  else if (prod < 0)
  {
    return -1;
  }
  else
  {
    return 0;
  }
}

float dist(pt_idx p1, pt_idx p2, pt_idx p)
{
  return abs((p.y - p1.y) * (p2.x - p1.x) - (p.x - p1.x) * (p2.y - p1.y));
}

__global__ void dist(pt_idx* p, float* d, int len, pt_idx p1, pt_idx p2)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < len)
  {
    pt_idx p3 = p[idx];
    d[idx] = abs((p3.y - p1.y) * (p2.x - p1.x) - (p3.x - p1.x) * (p2.y - p1.y));
  }
}

__global__ void max_dist(pt_idx* from_p, int from_len, pt_idx* to_p, int to_len, pt_idx p1, pt_idx p2)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float s_dist[BLOCK_SIZE];
  __shared__ pt_idx s_p[BLOCK_SIZE];
  s_dist[threadIdx.x] = 0;
  __syncthreads();

  if (idx < from_len)
  {
    s_p[threadIdx.x] = from_p[idx];
    pt_idx p3 = s_p[threadIdx.x];
    s_dist[threadIdx.x] = abs((p3.y - p1.y) * (p2.x - p1.x) - (p3.x - p1.x) * (p2.y - p1.y));
  }
  __syncthreads();

  // find max dist
  int s = blockDim.x / 2;
  while (s != 0)
  {
    if (threadIdx.x < s)
    {
      if (s_dist[threadIdx.x] < s_dist[threadIdx.x + s])
      {
        s_dist[threadIdx.x] = s_dist[threadIdx.x + s];
        s_p[threadIdx.x] = s_p[threadIdx.x + s];
      }
    }
    __syncthreads();
    s /= 2;
  }

  if (threadIdx.x == 0)
  {
    to_p[blockIdx.x] = s_p[0];
  }
}

__global__ void ccw(pt_idx* p, int* d, int len, pt_idx p1, pt_idx p2)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < len)
  {
    pt_idx p3 = p[idx];
    float prod = (p2.x - p1.x) * (p3.y - p1.y) - (p2.y - p1.y) * (p3.x - p1.x);
    // printf("p1 = (%f, %f), p2 = (%f, %f), p = (%f, %f), prod = %f\n", p1->x, p1->y, p2->x, p2->y, p3.x, p3.y, prod);
    if (prod > 0)
    {
      d[idx] = 1;
    }
    else if (prod < 0)
    {
      d[idx] = -1;
    }
    else
    {
      d[idx] = 0;
    }
  }
}

static void find_hull(vector<pt_idx>& pts, pt_idx p1, pt_idx p2, vector<unsigned>& indices)
{
  if (pts.size() == 0)
  {
    // printf("p1 = (%f, %f) idx = %u\n", p1->p.x, p1->p.y, p1->idx);
    indices.push_back(p1.idx);
    return;
  }

  vector<pt_idx> ac;
  vector<pt_idx> cb;
  auto p = pts[0];
  if (pts.size() > THRESHOLD)
  {
    pt_idx* d_p;
    pt_idx* d_p1;
    pt_idx* d_p2;
    hipMalloc(&d_p, sizeof(pt_idx) * pts.size());
    hipMalloc(&d_p1, sizeof(pt_idx) * pts.size());
    hipMalloc(&d_p2, sizeof(pt_idx) * pts.size());
    hipMemcpy(d_p, pts.data(), sizeof(pt_idx) * pts.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_p1, pts.data(), sizeof(pt_idx) * pts.size(), hipMemcpyHostToDevice);
    hipMemset(d_p2, 0, sizeof(pt_idx) * pts.size());

    int d_size = pts.size();
    while (d_size > 128)
    {
      //printf("d_size = %d\n", d_size);
      dim3 dimBlock(BLOCK_SIZE);
      int next_d_size = (d_size - 1) / BLOCK_SIZE + 1;
      dim3 dimGrid(next_d_size);
      max_dist<<<dimGrid, dimBlock>>>(d_p1, d_size, d_p2, next_d_size, p1, p2);
      pt_idx* temp = d_p1;
      d_p1 = d_p2;
      d_p2 = temp;
      d_size = next_d_size;
    }

    if (d_size > 1)
    {
      pt_idx *h_p;
      h_p = (pt_idx *)malloc(sizeof(pt_idx) * d_size);
      hipMemcpy(h_p, d_p1, sizeof(pt_idx) * d_size, hipMemcpyDeviceToHost);

      p = h_p[0];
      float max_dist = dist(p1, p2, p);
      for (int i = 1; i < d_size; i++)
      {
        float d = dist(p1, p2, h_p[i]);
        if (d > max_dist)
        {
          p = h_p[i];
          max_dist = d;
        }
      }
    }
    else
    {
      pt_idx *h_p;
      h_p = (pt_idx *)malloc(sizeof(pt_idx));
      hipMemcpy(h_p, d_p1, sizeof(pt_idx), hipMemcpyDeviceToHost);
      p = h_p[0];
    }

    int * d_ccw_ac;
    hipMalloc(&d_ccw_ac, sizeof(int) * pts.size());
    hipMemset(d_ccw_ac, 0, sizeof(int) * pts.size());

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid((pts.size() - 1) / BLOCK_SIZE + 1);
    ccw<<<dimGrid, dimBlock>>>(d_p, d_ccw_ac, pts.size(), p1, p);
    int * h_ccw_ac = (int *)malloc(sizeof(int) * pts.size());
    hipMemcpy(h_ccw_ac, d_ccw_ac, sizeof(int) * pts.size(), hipMemcpyDeviceToHost);

    int * d_ccw_cb;
    hipMalloc(&d_ccw_cb, sizeof(int) * pts.size());
    hipMemset(d_ccw_cb, 0, sizeof(int) * pts.size());

    ccw<<<dimGrid, dimBlock>>>(d_p, d_ccw_cb, pts.size(), p, p2);
    int * h_ccw_cb = (int *)malloc(sizeof(int) * pts.size());
    hipMemcpy(h_ccw_cb, d_ccw_cb, sizeof(int) * pts.size(), hipMemcpyDeviceToHost);

    for (int i = 0; i < pts.size(); i++)
    {
      if (pts[i] == p)
      {
        continue;
      }

      if (h_ccw_ac[i] == 1)
      {
        ac.push_back(pts[i]);
      }

      if (h_ccw_cb[i] == 1)
      {
        cb.push_back(pts[i]);
      }
    }

    hipFree(d_p);
    hipFree(d_p1);
    hipFree(d_p2);
    hipFree(d_ccw_ac);
    hipFree(d_ccw_cb);
  }
  else
  {
    float max_dist = dist(p1, p2, p);
    for (int i = 1; i < pts.size(); i++)
    {
      float d = dist(p1, p2, pts[i]);
      if (d > max_dist)
      {
        p = pts[i];
        max_dist = d;
      }
    }

    for (int i = 0; i < pts.size(); i++)
    {
      if (pts[i] == p)
      {
        continue;
      }

      int side = ccw(p1, p, pts[i]);
      if (side == 1)
      {
        ac.push_back(pts[i]);
      }

      side = ccw(p, p2, pts[i]);
      if (side == 1)
      {
        cb.push_back(pts[i]);
      }
    }
  }

  find_hull(ac, p1, p, indices);
  find_hull(cb, p, p2, indices);
}

static int compute(vector<pt_idx>& pts, vector<unsigned>& indices)
{
  auto left = pts[0];
  auto right = pts[pts.size() - 1];

  vector<pt_idx> ccw_pts;
  vector<pt_idx> cw_pts;

  if (pts.size() > THRESHOLD)
  {
    pt_idx* d_p;
    hipMalloc(&d_p, sizeof(pt_idx) * pts.size());
    hipMemcpy(d_p, pts.data(), sizeof(pt_idx) * pts.size(), hipMemcpyHostToDevice);

    int * d_ccw;
    hipMalloc(&d_ccw, sizeof(int) * pts.size());
    hipMemset(d_ccw, 0, sizeof(int) * pts.size());

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid((pts.size() - 1) / BLOCK_SIZE + 1);
    ccw<<<dimGrid, dimBlock>>>(d_p, d_ccw, pts.size(), left, right);

    int * h_ccw;
    h_ccw = (int *)malloc(sizeof(int) * pts.size());
    hipMemcpy(h_ccw, d_ccw, sizeof(int) * pts.size(), hipMemcpyDeviceToHost);

    hipFree(d_p);
    hipFree(d_ccw);

    for (int i = 1; i < pts.size() - 1; i++)
    {
      int side = h_ccw[i];
      if (side == 1)
      {
        // printf("pts[%d] = (%f, %f) idx = %u, side = 1\n", i, pts[i]->p.x, pts[i]->p.y, pts[i]->idx);
        ccw_pts.push_back(pts[i]);
      }
      else if (side == -1)
      {
        // printf("pts[%d] = (%f, %f) idx = %u, side = -1\n", i, pts[i]->p.x, pts[i]->p.y, pts[i]->idx);
        cw_pts.push_back(pts[i]);
      }
    }
  }
  else
  {
    for (int i = 1; i < pts.size() - 1; i++)
    {
      int side = ccw(left, right, pts[i]);
      if (side == 1)
      {
        // printf("pts[%d] = (%f, %f) idx = %u, side = 1\n", i, pts[i]->p.x, pts[i]->p.y, pts[i]->idx);
        ccw_pts.push_back(pts[i]);
      }
      else if (side == -1)
      {
        // printf("pts[%d] = (%f, %f) idx = %u, side = -1\n", i, pts[i]->p.x, pts[i]->p.y, pts[i]->idx);
        cw_pts.push_back(pts[i]);
      }
    }
  }

  find_hull(ccw_pts, left, right, indices);
  find_hull(cw_pts, right, left, indices);

  // for (int i = 0; i < indices.size(); i++)
  // {
  //   printf("indices[%d] = %u\n");
  // }
  return indices.size();
}

int main(int argc, char *argv[]) {
  gpuTKArg_t args;
  int inputLength;
  float *hostX;
  float *hostY;
  vector<pt_idx> hostPts;
  vector<unsigned> hostAnswer;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostX = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &inputLength);
  hostY = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 1), &inputLength);
  hostPts = vector<pt_idx>(inputLength);
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The input length is ", inputLength);

  gpuTKTime_start(Generic, "Total Computation");
  gpuTKTime_start(Generic, "Create data");
  for (unsigned i = 0; i < inputLength; i++)
  {
    hostPts[i] = pt_idx{i, hostX[i], hostY[i]};
  }

  thrust::device_vector<pt_idx> d_pts(hostPts);

  thrust::sort(d_pts.begin(), d_pts.end(), [] __host__ __device__ (const pt_idx& a, const pt_idx& b) {
    if (a.x < b.x) {
      return true;
    } else if (a.x > b.x) {
      return false;
    } else {
      return a.y < b.y;
    }
  });

  thrust::copy(d_pts.begin(), d_pts.end(), hostPts.begin());
  gpuTKTime_stop(Generic, "Create data");

  // Launch kernel
  // ----------------------------------------------------------
  gpuTKLog(TRACE, "Launching kernel");
  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Perform kernel computation here
  compute(hostPts, hostAnswer);
  gpuTKTime_stop(Compute, "Performing CUDA computation");
  gpuTKTime_stop(Generic, "Total Computation");
  // Verify correctness
  // -----------------------------------------------------
  gpuTKSolution(args, hostAnswer.data(), hostAnswer.size());

  // Free memory
  free(hostX);
  free(hostY);
  return 0;
}
