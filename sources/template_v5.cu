#include "hip/hip_runtime.h"
#include <gputk.h>
#include <algorithm>

#define BLOCK_SIZE 1024
#define THRESHOLD (BLOCK_SIZE * 32)

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

struct pt_idx {
  unsigned idx;
  float x, y;

  bool operator==(const pt_idx& rhs) const
  {
    return x == rhs.x && y == rhs.y && idx == rhs.idx;
  }
};

int ccw(pt_idx p1, pt_idx p2, pt_idx p)
{
  float prod = (p2.x - p1.x) * (p.y - p1.y) - (p2.y - p1.y) * (p.x - p1.x);
  if (prod > 0)
  {
    return 1;
  }
  else if (prod < 0)
  {
    return -1;
  }
  else
  {
    return 0;
  }
}

float dist(pt_idx p1, pt_idx p2, pt_idx p)
{
  return abs((p.y - p1.y) * (p2.x - p1.x) - (p.x - p1.x) * (p2.y - p1.y));
}

__global__ void dist(pt_idx* p, float* d, int len, pt_idx p1, pt_idx p2)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < len)
  {
    pt_idx p3 = p[idx];
    d[idx] = abs((p3.y - p1.y) * (p2.x - p1.x) - (p3.x - p1.x) * (p2.y - p1.y));
  }
}

__global__ void max_dist(pt_idx* from_p, int from_len, pt_idx* to_p, int to_len, pt_idx p1, pt_idx p2)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float s_dist[BLOCK_SIZE];
  __shared__ pt_idx s_p[BLOCK_SIZE];
  s_dist[threadIdx.x] = 0;
  __syncthreads();

  if (idx < from_len)
  {
    s_p[threadIdx.x] = from_p[idx];
    pt_idx p3 = s_p[threadIdx.x];
    s_dist[threadIdx.x] = abs((p3.y - p1.y) * (p2.x - p1.x) - (p3.x - p1.x) * (p2.y - p1.y));
  }
  __syncthreads();

  // find max dist
  int s = blockDim.x / 2;
  while (s != 0)
  {
    if (threadIdx.x < s)
    {
      if (s_dist[threadIdx.x] < s_dist[threadIdx.x + s])
      {
        s_dist[threadIdx.x] = s_dist[threadIdx.x + s];
        s_p[threadIdx.x] = s_p[threadIdx.x + s];
      }
    }
    __syncthreads();
    s /= 2;
  }

  if (threadIdx.x == 0)
  {
    to_p[blockIdx.x] = s_p[0];
  }
}

__global__ void ccw(pt_idx* p, int* d, int len, pt_idx p1, pt_idx p2)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < len)
  {
    pt_idx p3 = p[idx];
    float prod = (p2.x - p1.x) * (p3.y - p1.y) - (p2.y - p1.y) * (p3.x - p1.x);
    // printf("p1 = (%f, %f), p2 = (%f, %f), p = (%f, %f), prod = %f\n", p1->x, p1->y, p2->x, p2->y, p3.x, p3.y, prod);
    if (prod > 0)
    {
      d[idx] = 1;
    }
    else if (prod < 0)
    {
      d[idx] = -1;
    }
    else
    {
      d[idx] = 0;
    }
  }
}

static void find_hull(vector<pt_idx>& pts, pt_idx p1, pt_idx p2, vector<unsigned>& indices)
{
  if (pts.size() == 0)
  {
    // printf("p1 = (%f, %f) idx = %u\n", p1->p.x, p1->p.y, p1->idx);
    indices.push_back(p1.idx);
    return;
  }

  vector<pt_idx> ac;
  vector<pt_idx> cb;
  auto p = pts[0];
  if (pts.size() > THRESHOLD)
  {
    pt_idx* d_p;
    pt_idx* d_p1;
    pt_idx* d_p2;
    hipMallocManaged(&d_p, sizeof(pt_idx) * pts.size());
    hipMallocManaged(&d_p1, sizeof(pt_idx) * pts.size());
    hipMallocManaged(&d_p2, sizeof(pt_idx) * pts.size());

    memcpy(d_p, pts.data(), sizeof(pt_idx) * pts.size());
    memcpy(d_p1, pts.data(), sizeof(pt_idx) * pts.size());
    memset(d_p2, 0, sizeof(pt_idx) * pts.size());

    int d_size = pts.size();
    while (d_size > 128)
    {
      //printf("d_size = %d\n", d_size);
      dim3 dimBlock(BLOCK_SIZE);
      int next_d_size = (d_size - 1) / BLOCK_SIZE + 1;
      dim3 dimGrid(next_d_size);
      max_dist<<<dimGrid, dimBlock>>>(d_p1, d_size, d_p2, next_d_size, p1, p2);
      hipDeviceSynchronize();
      pt_idx* temp = d_p1;
      d_p1 = d_p2;
      d_p2 = temp;
      d_size = next_d_size;
    }

    if (d_size > 1)
    {
      p = d_p1[0];
      float max_dist = dist(p1, p2, p);
      for (int i = 1; i < d_size; i++)
      {
        float d = dist(p1, p2, d_p1[i]);
        if (d > max_dist)
        {
          p = d_p1[i];
          max_dist = d;
        }
      }
    }
    else
    {
      p = d_p1[0];
    }

    int * ccw_ac;
    hipMallocManaged(&ccw_ac, sizeof(int) * pts.size());

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid((pts.size() - 1) / BLOCK_SIZE + 1);
    ccw<<<dimGrid, dimBlock>>>(d_p, ccw_ac, pts.size(), p1, p);

    int * ccw_cb;
    hipMallocManaged(&ccw_cb, sizeof(int) * pts.size());

    ccw<<<dimGrid, dimBlock>>>(d_p, ccw_cb, pts.size(), p, p2);
    hipDeviceSynchronize();

    for (int i = 0; i < pts.size(); i++)
    {
      if (pts[i] == p)
      {
        continue;
      }

      if (ccw_ac[i] == 1)
      {
        ac.push_back(pts[i]);
      }

      if (ccw_cb[i] == 1)
      {
        cb.push_back(pts[i]);
      }
    }

    hipFree(d_p);
    hipFree(d_p1);
    hipFree(d_p2);
    hipFree(ccw_ac);
    hipFree(ccw_cb);
  }
  else
  {
    float max_dist = dist(p1, p2, p);
    for (int i = 1; i < pts.size(); i++)
    {
      float d = dist(p1, p2, pts[i]);
      if (d > max_dist)
      {
        p = pts[i];
        max_dist = d;
      }
    }

    for (int i = 0; i < pts.size(); i++)
    {
      if (pts[i] == p)
      {
        continue;
      }

      int side = ccw(p1, p, pts[i]);
      if (side == 1)
      {
        ac.push_back(pts[i]);
      }

      side = ccw(p, p2, pts[i]);
      if (side == 1)
      {
        cb.push_back(pts[i]);
      }
    }
  }

  find_hull(ac, p1, p, indices);
  find_hull(cb, p, p2, indices);
}

static int compute(vector<pt_idx>& pts, vector<unsigned>& indices)
{
  auto left = pts[0];
  auto right = pts[pts.size() - 1];

  vector<pt_idx> ccw_pts;
  vector<pt_idx> cw_pts;

  if (pts.size() > THRESHOLD)
  {
    pt_idx* p_uni;
    hipMallocManaged(&p_uni, sizeof(pt_idx) * pts.size());
    memcpy(p_uni, pts.data(), sizeof(pt_idx) * pts.size());

    int * ccw_uni;
    hipMallocManaged(&ccw_uni, sizeof(int) * pts.size());
    memcpy(ccw_uni, pts.data(), sizeof(int) * pts.size());

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid((pts.size() - 1) / BLOCK_SIZE + 1);
    ccw<<<dimGrid, dimBlock>>>(p_uni, ccw_uni, pts.size(), left, right);
    hipDeviceSynchronize();
    for (int i = 1; i < pts.size() - 1; i++)
    {
      int side = ccw_uni[i];
      if (side == 1)
      {
        // printf("pts[%d] = (%f, %f) idx = %u, side = 1\n", i, pts[i]->p.x, pts[i]->p.y, pts[i]->idx);
        ccw_pts.push_back(pts[i]);
      }
      else if (side == -1)
      {
        // printf("pts[%d] = (%f, %f) idx = %u, side = -1\n", i, pts[i]->p.x, pts[i]->p.y, pts[i]->idx);
        cw_pts.push_back(pts[i]);
      }
    }
    hipFree(p_uni);
    hipFree(ccw_uni);
  }
  else
  {
    for (int i = 1; i < pts.size() - 1; i++)
    {
      int side = ccw(left, right, pts[i]);
      if (side == 1)
      {
        // printf("pts[%d] = (%f, %f) idx = %u, side = 1\n", i, pts[i]->p.x, pts[i]->p.y, pts[i]->idx);
        ccw_pts.push_back(pts[i]);
      }
      else if (side == -1)
      {
        // printf("pts[%d] = (%f, %f) idx = %u, side = -1\n", i, pts[i]->p.x, pts[i]->p.y, pts[i]->idx);
        cw_pts.push_back(pts[i]);
      }
    }
  }

  find_hull(ccw_pts, left, right, indices);
  find_hull(cw_pts, right, left, indices);

  // for (int i = 0; i < indices.size(); i++)
  // {
  //   printf("indices[%d] = %u\n");
  // }
  return indices.size();
}

int main(int argc, char *argv[]) {
  gpuTKArg_t args;
  int inputLength;
  float *hostX;
  float *hostY;
  vector<pt_idx> hostPts;
  vector<unsigned> hostAnswer;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostX = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &inputLength);
  hostY = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 1), &inputLength);
  hostPts = vector<pt_idx>(inputLength);
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The input length is ", inputLength);

  gpuTKTime_start(Generic, "Total Computation");
  gpuTKTime_start(Generic, "Create data");
  for (unsigned i = 0; i < inputLength; i++)
  {
    hostPts[i] = pt_idx{i, hostX[i], hostY[i]};
  }
  std::sort(hostPts.begin(), hostPts.end(), [](const pt_idx& a, const pt_idx& b) {
    if (a.x < b.x) {
      return true;
    } else if (a.x > b.x) {
      return false;
    } else {
      return a.y < b.y;
    }
  });
  gpuTKTime_stop(Generic, "Create data");

  // Launch kernel
  // ----------------------------------------------------------
  gpuTKLog(TRACE, "Launching kernel");
  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Perform kernel computation here
  compute(hostPts, hostAnswer);
  gpuTKTime_stop(Compute, "Performing CUDA computation");
  gpuTKTime_stop(Generic, "Total Computation");

  // Verify correctness
  // -----------------------------------------------------
  gpuTKSolution(args, hostAnswer.data(), hostAnswer.size());

  // Free memory
  free(hostX);
  free(hostY);
  return 0;
}
